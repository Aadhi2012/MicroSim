#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stddef.h>
#include <stdlib.h>
#include <math.h>
#include <sys/stat.h>
#include <fenv.h>
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>
#include <hipfft/hipfft.h>
#include <hipblas.h>
#include <hip/hip_math_constants.h>

#include "functions/helper_string.h"
#include "functions/hip/hip_runtime_api.h"
#include "hipcub/hipcub.hpp"

#include "functions/kernels.cuh"
#include "functions/global_vars.h"
#include "functions/utility.h"
#include "functions/functions.h"
#include "functions/filling.h"
#include "functions/cuda_filling.cuh"
#include "functions/reading_input_parameters.h"
#include "functions/initialize_variables.h"
#include "functions/fill_domain.h"
#include "functions/calc_bn.h"

#include "solverloop/file_writer.h"
#include "solverloop/evolve.h"

int main (int argc, char *argv[]) {

    // Check to ensure the program is run with all the necessary info specified
    if (argc < 4)
    {
        printf("Insufficient number of arguments in execution command\n"
        "Using default arguments - Input.in, Filling.in and Output\n");

        const int argc_const = 4;
        argc = argc_const;

        char *argv_temp[argc_const];

        for (int i = 0; i < argc; i++)
            argv_temp[i] = (char*)malloc(20 * sizeof(char));

        strcpy(argv_temp[0], "kks.out");
        strcpy(argv_temp[1], "Input.in");
        strcpy(argv_temp[2], "Filling.in");
        strcpy(argv_temp[3], "Output");

        for (int i = 0; i < argc; i++)
            argv[i] = argv_temp[i];
    }

    mkdir("DATA",0777);

    // Get CUDA device properties
    devPropQuery();

    // Reads all the input data from the specified input file
    reading_input_parameters(argv);

    printf("Periodic boundary conditions will be applied for all variables at all boundaries\n");

    // Uses the information read by reading_input_parameters to allocate memory
    // for variables that are necessary for the simulation
    initialize_variables();

    // Sets the initial phase-fields using the geometric data
    // specified in the filling file
    fill_domain(argv);

    // Executes the solution procedure
    if (DIMENSION == 3)
        Evolve3D(argv);
    else if (DIMENSION == 2)
        Evolve2D(argv);

    hipDeviceReset();
    return EXIT_SUCCESS;
}
