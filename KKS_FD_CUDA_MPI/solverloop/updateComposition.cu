#include "hip/hip_runtime.h"
#include "updateComposition.cuh"

__global__
void __updateComposition__(double **phi,
                           double **comp, double **compNew,
                           double **phaseComp,
                           double *F0_A, double *F0_B,
                           double *mobility,
                           long NUMPHASES, long NUMCOMPONENTS, long DIMENSION,
                           long sizeX, long sizeY, long sizeZ,
                           long yStep, long zStep, long padding,
                           double DELTA_X, double DELTA_Y, double DELTA_Z,
                           double DELTA_t)
{
    long i = threadIdx.x + blockIdx.x * blockDim.x;
    long j = threadIdx.y + blockIdx.y * blockDim.y;
    long k = threadIdx.z + blockIdx.z * blockDim.z;

    long idx[7] = {-1};

    idx[0] = k*zStep + j*yStep + i;

    double mu[7];
    double effMobility[7];
    double J_xp = 0.0, J_xm = 0.0, J_yp = 0.0, J_ym = 0.0, J_zp = 0.0, J_zm = 0.0;

    if (i >= padding && i < sizeX-padding && ((j >= padding && j < sizeY-padding && DIMENSION >= 2) || (DIMENSION == 1 && j == 0)) && ((k >= padding && k < sizeZ-padding && DIMENSION == 3) || (DIMENSION < 3 && k == 0)))
    {
        // x-direction
        idx[1] = k*zStep + j*yStep + i+1;
        idx[2] = k*zStep + j*yStep + i-1;

        // y-direction
        if (DIMENSION >= 2)
        {
            idx[3] = k*zStep + (j+1)*yStep + i;
            idx[4] = k*zStep + (j-1)*yStep + i;
        }

        // z-direction
        if (DIMENSION == 3)
        {
            idx[5] = (k+1)*zStep + j*yStep + i;
            idx[6] = (k-1)*zStep + j*yStep + i;
        }

        for (long component = 0; component < NUMCOMPONENTS-1; component++)
        {
            J_xp = 0.0;
            J_xm = 0.0;
            J_yp = 0.0;
            J_ym = 0.0;
            J_zp = 0.0;
            J_zm = 0.0;

            for (long component2 = 0; component2 < NUMCOMPONENTS-1; component2++)
            {

                for (long iter = 0; iter < 7; iter++)
                    effMobility[iter] = 0.0;

                for (long phase = 0; phase < NUMPHASES; phase++)
                {
                    effMobility[0] += mobility[(component2 + phase*(NUMCOMPONENTS-1))*(NUMCOMPONENTS-1) + component]*calcInterp5th(phi, phase, idx[0], NUMPHASES);

                    effMobility[1] += mobility[(component2 + phase*(NUMCOMPONENTS-1))*(NUMCOMPONENTS-1) + component]*calcInterp5th(phi, phase, idx[1], NUMPHASES);
                    effMobility[2] += mobility[(component2 + phase*(NUMCOMPONENTS-1))*(NUMCOMPONENTS-1) + component]*calcInterp5th(phi, phase, idx[2], NUMPHASES);

                    if (DIMENSION >= 2)
                    {
                        effMobility[3] += mobility[(component2 + phase*(NUMCOMPONENTS-1))*(NUMCOMPONENTS-1) + component]*calcInterp5th(phi, phase, idx[3], NUMPHASES);
                        effMobility[4] += mobility[(component2 + phase*(NUMCOMPONENTS-1))*(NUMCOMPONENTS-1) + component]*calcInterp5th(phi, phase, idx[4], NUMPHASES);
                    }

                    if (DIMENSION == 3)
                    {
                        effMobility[5] += mobility[(component2 + phase*(NUMCOMPONENTS-1))*(NUMCOMPONENTS-1) + component]*calcInterp5th(phi, phase, idx[5], NUMPHASES);
                        effMobility[6] += mobility[(component2 + phase*(NUMCOMPONENTS-1))*(NUMCOMPONENTS-1) + component]*calcInterp5th(phi, phase, idx[6], NUMPHASES);
                    }
                }


                mu[0] = calcDiffusionPotential(phaseComp, NUMPHASES-1, component2, F0_A, F0_B, idx[0], NUMPHASES, NUMCOMPONENTS);

                mu[1] = calcDiffusionPotential(phaseComp, NUMPHASES-1, component2, F0_A, F0_B, idx[1], NUMPHASES, NUMCOMPONENTS);
                mu[2] = calcDiffusionPotential(phaseComp, NUMPHASES-1, component2, F0_A, F0_B, idx[2], NUMPHASES, NUMCOMPONENTS);

                if (DIMENSION >= 2)
                {
                    mu[3] = calcDiffusionPotential(phaseComp, NUMPHASES-1, component2, F0_A, F0_B, idx[3], NUMPHASES, NUMCOMPONENTS);
                    mu[4] = calcDiffusionPotential(phaseComp, NUMPHASES-1, component2, F0_A, F0_B, idx[4], NUMPHASES, NUMCOMPONENTS);
                }

                if (DIMENSION == 3)
                {
                    mu[5] = calcDiffusionPotential(phaseComp, NUMPHASES-1, component2, F0_A, F0_B, idx[5], NUMPHASES, NUMCOMPONENTS);
                    mu[6] = calcDiffusionPotential(phaseComp, NUMPHASES-1, component2, F0_A, F0_B, idx[6], NUMPHASES, NUMCOMPONENTS);
                }

                J_xp += ((effMobility[1] + effMobility[0])/2.0)*(mu[1] - mu[0])/DELTA_X;
                J_xm += ((effMobility[0] + effMobility[2])/2.0)*(mu[0] - mu[2])/DELTA_X;

                if (DIMENSION >= 2)
                {
                    J_yp += ((effMobility[3] + effMobility[0])/2.0)*(mu[3] - mu[0])/DELTA_Y;
                    J_ym += ((effMobility[0] + effMobility[4])/2.0)*(mu[0] - mu[4])/DELTA_Y;
                }

                if (DIMENSION == 3)
                {
                    J_zp += ((effMobility[5] + effMobility[0])/2.0)*(mu[5] - mu[0])/DELTA_Z;
                    J_zm += ((effMobility[0] + effMobility[6])/2.0)*(mu[0] - mu[6])/DELTA_Z;
                }
            }

            compNew[component][idx[0]] = comp[component][idx[0]] + DELTA_t*((J_xp - J_xm)/DELTA_X + (J_yp - J_ym)/DELTA_Y + (J_zp - J_zm)/DELTA_Z);
        }
    }
}

__global__
void __updateComposition_02__(double **phi,
                              double **comp, double **compNew, double **mu,
                              double **phaseComp, long *thermo_phase,
                              double *diffusivity, double temperature, double molarVolume,
                              long NUMPHASES, long NUMCOMPONENTS, long DIMENSION,
                              long sizeX, long sizeY, long sizeZ,
                              long yStep, long zStep, long padding,
                              double DELTA_X, double DELTA_Y, double DELTA_Z,
                              double DELTA_t)
{
    long i = threadIdx.x + blockIdx.x * blockDim.x;
    long j = threadIdx.y + blockIdx.y * blockDim.y;
    long k = threadIdx.z + blockIdx.z * blockDim.z;

    long idx[7] = {-1};

    idx[0] = k*zStep + j*yStep + i;

    double muLocal[7];
    double effMobility[7];
    double J_xp = 0.0, J_xm = 0.0, J_yp = 0.0, J_ym = 0.0, J_zp = 0.0, J_zm = 0.0;
    double tol = 1e-6;

    if (i >= padding && i < sizeX-padding && ((j >= padding && j < sizeY-padding && DIMENSION >= 2) || (DIMENSION == 1 && j == 0)) && ((k >= padding && k < sizeZ-padding && DIMENSION == 3) || (DIMENSION < 3 && k == 0)))
    {
        // x-direction
        idx[1] = k*zStep + j*yStep + i+1;
        idx[2] = k*zStep + j*yStep + i-1;

        // y-direction
        if (DIMENSION >= 2)
        {
            idx[3] = k*zStep + (j+1)*yStep + i;
            idx[4] = k*zStep + (j-1)*yStep + i;
        }

        // z-direction
        if (DIMENSION == 3)
        {
            idx[5] = (k+1)*zStep + j*yStep + i;
            idx[6] = (k-1)*zStep + j*yStep + i;
        }

        double dmudc[(MAX_NUM_COMP)*(MAX_NUM_COMP)];
        double y[MAX_NUM_COMP];
        double dmudcInv[MAX_NUM_COMP][MAX_NUM_COMP];
        int P[MAX_NUM_COMP];
        double mobility[MAX_NUM_COMP][MAX_NUM_COMP];

        long maxPos;

        long interface = 1, bulkphase = 0;

        for (long is = 0; is < NUMPHASES; is++)
        {
            if (phi[is][idx[0]] > 0.99999)
            {
                bulkphase = is;
                interface = 0;
                break;
            }
        }

        if (interface)
        {
            for (int component = 0; component < NUMCOMPONENTS-1; component++)
            {
                // Fluxes
                J_xp = 0.0;
                J_xm = 0.0;
                J_yp = 0.0;
                J_ym = 0.0;
                J_zp = 0.0;
                J_zm = 0.0;

                // Computing the inner derivative and mobilities to get the fluxes
                for (long component2 = 0; component2 < NUMCOMPONENTS-1; component2++)
                {
                    for (long iter = 0; iter < 7; iter++)
                        effMobility[iter] = 0.0;

                    if (DIMENSION == 3)
                        maxPos = 7;
                    else if (DIMENSION == 2)
                        maxPos = 5;
                    else
                        maxPos = 3;

                    for (long pos = 0; pos < maxPos; pos++)
                    {
                        // M_{ij} = \sum_{\phi} M(\phi) = \sum_{\phi} D*dcdmu
                        for (long phase = 0; phase < NUMPHASES; phase++)
                        {
                            double tmp0 = 0.0;

                            for (long is = 0; is < NUMCOMPONENTS-1; is++)
                            {
                                y[is] = phaseComp[is*NUMPHASES + phase][idx[pos]];
                                tmp0  += y[is];
                            }

                            y[NUMCOMPONENTS-1] = 1.0 - tmp0;


                            // Get dmudc for the current phase
                            (*dmudc_tdb_dev[thermo_phase[phase]])(temperature, y, dmudc);

                            // Invert dmudc to get dcdmu for the current phase
                            LUPDecomposeC2(dmudc, NUMCOMPONENTS-1, tol, P);
                            LUPInvertC2(dmudc, P, NUMCOMPONENTS-1, dmudcInv);

                            // multiply diffusivity with dcdmu
                            for (long iter1 = 0; iter1 < NUMCOMPONENTS-1; iter1++)
                            {
                                for (long iter2 = 0; iter2 < NUMCOMPONENTS-1; iter2++)
                                {
                                    mobility[iter1][iter2] = 0.0;

                                    for (long iter3 = 0; iter3 < NUMCOMPONENTS-1; iter3++)
                                    {
                                        mobility[iter1][iter2] += diffusivity[(iter1 + phase*(NUMCOMPONENTS-1))*(NUMCOMPONENTS-1) + iter3]*dmudcInv[iter3][iter2];
                                    }
                                }
                            }

                            // Summing over all phases, weighting with the interpolation fn.
                            effMobility[pos] += mobility[component][component2]*calcInterp5th(phi, phase, idx[pos], NUMPHASES);
                        }
                    }


                    muLocal[0] = mu[component2][idx[0]];

                    muLocal[1] = mu[component2][idx[1]];
                    muLocal[2] = mu[component2][idx[2]];

                    if (DIMENSION >= 2)
                    {
                        muLocal[3] = mu[component2][idx[3]];
                        muLocal[4] = mu[component2][idx[4]];
                    }
                    if (DIMENSION == 3)
                    {
                        muLocal[5] = mu[component2][idx[5]];
                        muLocal[6] = mu[component2][idx[6]];
                    }

                    J_xp += ((effMobility[1] + effMobility[0])/2.0)*(muLocal[1] - muLocal[0])/DELTA_X;
                    J_xm += ((effMobility[0] + effMobility[2])/2.0)*(muLocal[0] - muLocal[2])/DELTA_X;

                    if (DIMENSION >= 2)
                    {
                        J_yp += ((effMobility[3] + effMobility[0])/2.0)*(muLocal[3] - muLocal[0])/DELTA_Y;
                        J_ym += ((effMobility[0] + effMobility[4])/2.0)*(muLocal[0] - muLocal[4])/DELTA_Y;
                    }

                    if (DIMENSION == 3)
                    {
                        J_zp += ((effMobility[5] + effMobility[0])/2.0)*(muLocal[5] - muLocal[0])/DELTA_Z;
                        J_zm += ((effMobility[0] + effMobility[6])/2.0)*(muLocal[0] - muLocal[6])/DELTA_Z;
                    }
                }

                compNew[component][idx[0]] = comp[component][idx[0]] + DELTA_t*((J_xp - J_xm)/DELTA_X + (J_yp - J_ym)/DELTA_Y + (J_zp - J_zm)/DELTA_Z);
            }
        }
        else
        {
            for (int component = 0; component < NUMCOMPONENTS-1; component++)
            {
                // Fluxes
                J_xp = 0.0;
                J_xm = 0.0;
                J_yp = 0.0;
                J_ym = 0.0;
                J_zp = 0.0;
                J_zm = 0.0;

                // Computing the inner derivative and mobilities to get the fluxes
                for (long component2 = 0; component2 < NUMCOMPONENTS-1; component2++)
                {
                    for (long iter = 0; iter < 7; iter++)
                        effMobility[iter] = 0.0;

                    if (DIMENSION == 3)
                        maxPos = 7;
                    else if (DIMENSION == 2)
                        maxPos = 5;
                    else
                        maxPos = 3;

                    for (long pos = 0; pos < maxPos; pos++)
                    {
                        // M_{ij} = \sum_{\phi} M(\phi) = \sum_{\phi} D*dcdmu
                        double tmp0 = 0.0;

                        for (long is = 0; is < NUMCOMPONENTS-1; is++)
                        {
                            y[is] = phaseComp[is*NUMPHASES + bulkphase][idx[pos]];
                            tmp0  += y[is];
                        }

                        y[NUMCOMPONENTS-1] = 1.0 - tmp0;


                        // Get dmudc for the current phase
                        (*dmudc_tdb_dev[thermo_phase[bulkphase]])(temperature, y, dmudc);

                        // Invert dmudc to get dcdmu for the current phase
                        LUPDecomposeC2(dmudc, NUMCOMPONENTS-1, tol, P);
                        LUPInvertC2(dmudc, P, NUMCOMPONENTS-1, dmudcInv);

                        // multiply diffusivity with dcdmu
                        for (long iter1 = 0; iter1 < NUMCOMPONENTS-1; iter1++)
                        {
                            for (long iter2 = 0; iter2 < NUMCOMPONENTS-1; iter2++)
                            {
                                mobility[iter1][iter2] = 0.0;

                                for (long iter3 = 0; iter3 < NUMCOMPONENTS-1; iter3++)
                                {
                                    mobility[iter1][iter2] += diffusivity[(iter1 + bulkphase*(NUMCOMPONENTS-1))*(NUMCOMPONENTS-1) + iter3]*dmudcInv[iter3][iter2];
                                }
                            }
                        }

                        // Summing over all phases, weighting with the interpolation fn.
                        effMobility[pos] += mobility[component][component2]*calcInterp5th(phi, bulkphase, idx[pos], NUMPHASES);
                    }


                    muLocal[0] = mu[component2][idx[0]];

                    muLocal[1] = mu[component2][idx[1]];
                    muLocal[2] = mu[component2][idx[2]];

                    if (DIMENSION >= 2)
                    {
                        muLocal[3] = mu[component2][idx[3]];
                        muLocal[4] = mu[component2][idx[4]];
                    }
                    if (DIMENSION == 3)
                    {
                        muLocal[5] = mu[component2][idx[5]];
                        muLocal[6] = mu[component2][idx[6]];
                    }

                    J_xp += ((effMobility[1] + effMobility[0])/2.0)*(muLocal[1] - muLocal[0])/DELTA_X;
                    J_xm += ((effMobility[0] + effMobility[2])/2.0)*(muLocal[0] - muLocal[2])/DELTA_X;

                    if (DIMENSION >= 2)
                    {
                        J_yp += ((effMobility[3] + effMobility[0])/2.0)*(muLocal[3] - muLocal[0])/DELTA_Y;
                        J_ym += ((effMobility[0] + effMobility[4])/2.0)*(muLocal[0] - muLocal[4])/DELTA_Y;
                    }

                    if (DIMENSION == 3)
                    {
                        J_zp += ((effMobility[5] + effMobility[0])/2.0)*(muLocal[5] - muLocal[0])/DELTA_Z;
                        J_zm += ((effMobility[0] + effMobility[6])/2.0)*(muLocal[0] - muLocal[6])/DELTA_Z;
                    }
                }

                compNew[component][idx[0]] = comp[component][idx[0]] + DELTA_t*((J_xp - J_xm)/DELTA_X + (J_yp - J_ym)/DELTA_Y + (J_zp - J_zm)/DELTA_Z);
            }
        }
    }
}

__global__
void __updateMu_02__(double **phi, double **comp,
                     double **phiNew, double **compNew,
                     double **phaseComp, double **mu,
                     long *thermo_phase, double temperature, double molarVolume,
                     long NUMPHASES, long NUMCOMPONENTS, long DIMENSION,
                     long sizeX, long sizeY, long sizeZ,
                     long yStep, long zStep, long padding,
                     double DELTA_X, double DELTA_Y, double DELTA_Z,
                     double DELTA_t)
{
    long i = threadIdx.x + blockIdx.x * blockDim.x;
    long j = threadIdx.y + blockIdx.y * blockDim.y;
    long k = threadIdx.z + blockIdx.z * blockDim.z;

    long idx = k*zStep + j*yStep + i;

    if (i >= padding && i < sizeX-padding && ((j >= padding && j < sizeY-padding && DIMENSION >= 2) || (DIMENSION == 1 && j == 0)) && ((k >= padding && k < sizeZ-padding && DIMENSION == 3) || (DIMENSION < 3 && k == 0)))
    {
        double RHS[MAX_NUM_COMP] = {0.0}, sum = 0.0;
        double tol = 1e-6;

        long bulkphase = 0, interface = 1;

        for (long is = 0; is < NUMPHASES; is++)
        {
            if (phi[is][idx] > 0.99999)
            {
                bulkphase = is;
                interface = 0;
                break;
            }
        }

        if (interface)
        {
            double dmudc[(MAX_NUM_COMP)*(MAX_NUM_COMP)];
            double dcdmu[(MAX_NUM_COMP)*(MAX_NUM_COMP)];
            double y[MAX_NUM_COMP];
            double Inv[MAX_NUM_COMP][MAX_NUM_COMP];
            int P[MAX_NUM_COMP];

            for (long component = 0; component < NUMCOMPONENTS-1; component++)
            {
                for (long component2 = 0; component2 < NUMCOMPONENTS-1; component2++)
                {
                    dcdmu[component*(NUMCOMPONENTS-1) + component2] = 0.0;
                }
            }

            for (long phase = 0; phase < NUMPHASES; phase++)
            {
                sum = 0.0;

                for (long component = 0; component < NUMCOMPONENTS-1; component++)
                {
                    y[component] = phaseComp[component*NUMPHASES + phase][idx];
                    sum += y[component];
                }

                y[NUMCOMPONENTS-1] = 1.0 - sum;

                (*dmudc_tdb_dev[thermo_phase[phase]])(temperature, y, dmudc);

                LUPDecomposeC2(dmudc, NUMCOMPONENTS-1, tol, P);
                LUPInvertC2(dmudc, P, NUMCOMPONENTS-1, Inv);

                for (long component = 0; component < NUMCOMPONENTS-1; component++)
                    for (long component2 = 0; component2 < NUMCOMPONENTS-1; component2++)
                        dcdmu[component*(NUMCOMPONENTS-1) + component2] += calcInterp5th(phi, phase, idx, NUMPHASES)*Inv[component][component2];
            }

            LUPDecomposeC2(dcdmu, NUMCOMPONENTS-1, tol, P);
            LUPInvertC2(dcdmu, P, NUMCOMPONENTS-1, Inv);

            for (long component = 0; component < NUMCOMPONENTS-1; component++)
            {
                RHS[component] = (compNew[component][idx] - comp[component][idx]);

                for (long phase = 0; phase < NUMPHASES; phase++)
                {
                    sum = 0.0;

                    for (long phase2 = 0; phase2 < NUMPHASES; phase2++)
                    {
                        sum += calcInterp5thDiff(phi, phase, phase2, idx, NUMPHASES)*(phiNew[phase2][idx] - phi[phase2][idx]);
                    }

                    RHS[component] -= phaseComp[phase + component*NUMPHASES][idx]*sum;
                }
            }

            for (long component = 0; component < NUMCOMPONENTS-1; component++)
            {
                for (long component2 = 0; component2 < NUMCOMPONENTS-1; component2++)
                {
                    mu[component][idx] += Inv[component][component2]*RHS[component2];
                }
            }
        }
        else
        {
            double y[MAX_NUM_COMP];
            double mu1[MAX_NUM_COMP];

            sum = 0.0;

            for (long is = 0; is < NUMCOMPONENTS-1; is++)
            {
                y[is] = compNew[is][idx];
                sum += y[is];
            }

            y[NUMCOMPONENTS-1] = 1.0 - sum;

            (*Mu_tdb_dev[thermo_phase[bulkphase]])(temperature, y, mu1);

            for (long is = 0; is < NUMCOMPONENTS-1; is++)
                mu[is][idx] = mu1[is];
        }
    }
}

void updateComposition(double **phi, double **comp, double **phiNew, double **compNew,
                       double **phaseComp, double **mu,
                       domainInfo* simDomain, controls* simControls,
                       simParameters* simParams, subdomainInfo* subdomain,
                       dim3 gridSize, dim3 blockSize)
{
    if (simControls->FUNCTION_F == 1 || simControls->FUNCTION_F == 3 || simControls->FUNCTION_F == 4)
    {
        __updateComposition__<<<gridSize, blockSize>>>(phi, comp, compNew,
                                                       phaseComp,
                                                       simParams->F0_A_dev, simParams->F0_B_dev,
                                                       simParams->mobility_dev,
                                                       simDomain->numPhases, simDomain->numComponents, simDomain->DIMENSION,
                                                       subdomain->sizeX, subdomain->sizeY, subdomain->sizeZ,
                                                       subdomain->yStep, subdomain->zStep, subdomain->padding,
                                                       simDomain->DELTA_X, simDomain->DELTA_Y, simDomain->DELTA_Z,
                                                       simControls->DELTA_t);

        applyBoundaryCondition(compNew, 2, simDomain->numComponents-1,
                               simDomain, simControls,
                               simParams, subdomain,
                               gridSize, blockSize);
    }
    else if (simControls->FUNCTION_F == 2)
    {
        __updateComposition_02__<<<gridSize, blockSize>>>(phi, comp,
                                                          compNew, mu,
                                                          phaseComp, simDomain->thermo_phase_dev,
                                                          simParams->diffusivity_dev, simParams->T, simParams->molarVolume,
                                                          simDomain->numPhases, simDomain->numComponents, simDomain->DIMENSION,
                                                          subdomain->sizeX, subdomain->sizeY, subdomain->sizeZ,
                                                          subdomain->yStep, subdomain->zStep, subdomain->padding,
                                                          simDomain->DELTA_X, simDomain->DELTA_Y, simDomain->DELTA_Z,
                                                          simControls->DELTA_t);

        applyBoundaryCondition(compNew, 2, simDomain->numComponents-1,
                               simDomain, simControls,
                               simParams, subdomain,
                               gridSize, blockSize);

        __updateMu_02__<<<gridSize, blockSize>>>(phi, comp,
                                                 phiNew, compNew,
                                                 phaseComp, mu,
                                                 simDomain->thermo_phase_dev, simParams->T, simParams->molarVolume,
                                                 simDomain->numPhases, simDomain->numComponents, simDomain->DIMENSION,
                                                 subdomain->sizeX, subdomain->sizeY, subdomain->sizeZ,
                                                 subdomain->yStep, subdomain->zStep, subdomain->padding,
                                                 simDomain->DELTA_X, simDomain->DELTA_Y, simDomain->DELTA_Z,
                                                 simControls->DELTA_t);

        applyBoundaryCondition(mu, 1, simDomain->numComponents-1,
                               simDomain, simControls,
                               simParams, subdomain,
                               gridSize, blockSize);

    }
}
