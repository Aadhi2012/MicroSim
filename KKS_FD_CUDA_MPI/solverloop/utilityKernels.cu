#include "hip/hip_runtime.h"
#include "utilityKernels.cuh"
#include <cub.cuh>

__device__ __host__
double evalFunc(void f(double, double*, double*), double x, double temperature)
{
    double c[2];
    c[0] = x;
    c[1] = 1.0 - c[0];

    double ans;

    f(temperature, c, &ans);

    // Non-dimensionalise
    ans /= (1.602*1e8);

    return ans;
}

__device__ __host__
double spline_eval(double x, double *controlPoints,
                   double *a, double *b, double *c, double *d,
                   int numControlPoints)
{
    double ans = 0.0;

    for (int i = 0; i < numControlPoints-1; i++)
    {
        if (x >= controlPoints[i] && x <= controlPoints[i+1])
        {
            ans = x - controlPoints[i];
            ans = d[i] + ans*(c[i] + ans*(b[i] + ans*a[i]));
        }
    }

    return ans;
}

__global__
void computeChange(double *A, double *B,
                   int sizeX, int sizeY, int sizeZ)
{
    int i = threadIdx.x + blockIdx.x*blockDim.x;
    int j = threadIdx.y + blockIdx.y*blockDim.y;
    int k = threadIdx.z + blockIdx.z*blockDim.z;

    int idx = (j + k*sizeY)*sizeX + i;

    if (i < sizeX && j < sizeY && k < sizeZ)
    {
        A[idx] = fabs(B[idx] - A[idx]);
    }
    __syncthreads();
}

void printStats(double **phi, double **comp,
                double **phiNew, double **compNew,
                double *maxerr, double *maxVal, double *minVal,
                domainInfo simDomain, subdomainInfo subdomain,
                dim3 gridSize, dim3 blockSize)
{
    int i, j = 0;

    void    *t_storage = NULL;
    size_t  t_storage_bytes = 0;

    double *temp;
    hipMalloc((void**)&temp, sizeof(double));

    hipcub::DeviceReduce::Max(t_storage, t_storage_bytes, phiNew[1], temp, subdomain.numCells);
    hipcub::DeviceReduce::Sum(t_storage, t_storage_bytes, phiNew[1], temp, subdomain.numCells);
    hipMalloc((void**)&t_storage, t_storage_bytes);

    for (i = 0; i < simDomain.numComponents-1; i++)
    {
        hipcub::DeviceReduce::Max(t_storage, t_storage_bytes, compNew[i] + subdomain.shiftPointer, temp, subdomain.numCells);
        hipMemcpy(maxVal+j, temp, sizeof(double), hipMemcpyDeviceToDevice);
        hipcub::DeviceReduce::Min(t_storage, t_storage_bytes, compNew[i] + subdomain.shiftPointer, temp, subdomain.numCells);
        hipMemcpy(minVal+j, temp, sizeof(double), hipMemcpyDeviceToDevice);

        computeChange<<<gridSize, blockSize>>>(comp[i], compNew[i], subdomain.sizeX, subdomain.sizeY, subdomain.sizeZ);

        hipcub::DeviceReduce::Max(t_storage, t_storage_bytes, comp[i] + subdomain.shiftPointer, temp, subdomain.numCells);
        hipMemcpy(maxerr+j, temp, sizeof(double), hipMemcpyDeviceToDevice);

        j++;
    }

    for (i = 0; i < simDomain.numPhases; i++)
    {
        hipcub::DeviceReduce::Max(t_storage, t_storage_bytes, phiNew[i] + subdomain.shiftPointer, temp, subdomain.numCells);
        hipMemcpy(maxVal+j, temp, sizeof(double), hipMemcpyDeviceToDevice);
        hipcub::DeviceReduce::Min(t_storage, t_storage_bytes, phiNew[i] + subdomain.shiftPointer, temp, subdomain.numCells);
        hipMemcpy(minVal+j, temp, sizeof(double), hipMemcpyDeviceToDevice);

        computeChange<<<gridSize, blockSize>>>(phi[i], phiNew[i], subdomain.sizeX, subdomain.sizeY, subdomain.sizeZ);

        hipcub::DeviceReduce::Max(t_storage, t_storage_bytes, phi[i] + subdomain.shiftPointer, temp, subdomain.numCells);
        hipMemcpy(maxerr+j, temp, sizeof(double), hipMemcpyDeviceToDevice);

        j++;
    }

    hipFree(temp);
    hipFree(t_storage);
}
